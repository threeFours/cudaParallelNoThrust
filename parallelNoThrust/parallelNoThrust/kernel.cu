#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define vectorSize 1000

float totalTime;

hipError_t parallelWithCuda(double *a, double *b, double *c, unsigned int size);

__global__ void parallelVM(double *a, double *b, double *c)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	c[i] = a[i] * b[i];
}

int main()
{
    int i;
	double k = 5.0;
	double j = (float)vectorSize/2.0;

	double a[vectorSize];
	double b[vectorSize]; 
	double c[vectorSize]; 

	// fill vectors with random values
	for ( i = 0; i < vectorSize; i++ ){
				a[i] = (double)rand()/ (double)RAND_MAX;
				b[i] = (double)rand()/ (double)RAND_MAX;
				c[i] = 0.0;
	}

    // Multiply vectors
    hipError_t cudaStatus = parallelWithCuda(a, b, c, vectorSize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "seqWithCuda failed!");
        return 1;
    }
	/*
	// display vectors for debugging
	for ( i = 0; i < vectorSize; i++ ){
				printf("\n%f * %f = %f", a[i], b[i], c[i]);
	}
	*/

	printf("\n\nTotal time: %.3fms\n\n", totalTime);
	
    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipDeviceReset();

    return 0;
}

// Helper function for using CUDA to multiply vectors
hipError_t parallelWithCuda(double *a, double *b, double *c, unsigned int size)
{
    double *dev_a;
    double *dev_b;
    double *dev_c;
    hipError_t cudaStatus;
	hipError_t error;
	hipEvent_t start, stop;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto End;
    }

    // Allocate GPU buffers for three vectors (two input, one output)  

    hipMalloc((void**)&dev_a, size * sizeof(double));
	hipMalloc((void**)&dev_b, size * sizeof(double));
	hipMalloc((void**)&dev_c, size * sizeof(double));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(double), hipMemcpyHostToDevice);

	// Allocate CUDA events that we'll use for timing
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, NULL);

	// multiply the vectors
	parallelVM<<< vectorSize/512, 512 >>>(dev_a, dev_b, dev_c);

	// Record the stop event
    hipEventRecord(stop, NULL);
    // Wait for the stop event to complete
    hipEventSynchronize(stop);

	totalTime = 0.0f;
	hipEventElapsedTime(&totalTime, start, stop);

    hipDeviceSynchronize();

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(c, dev_c, size * sizeof(double), hipMemcpyDeviceToHost);

End:
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    
    return cudaStatus;
}
